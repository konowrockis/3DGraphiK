#include "hip/hip_runtime.h"
#include "GL/glew.h"
#include <hip/hip_runtime.h>
#include <>
#include "rasterizer.h"
#include "VectorUtils.h"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cuda_gl_interop.h>

#include "GLM/vec3.hpp"
#include "GLM/vec4.hpp"
#include "GLM/mat4x4.hpp"

#include "stream_compaction.h"

#define VERTEX_SHADER_BLOCK_SIZE 256
#define RASTERIZER_BLOCK_SIZE 9
#define FRAMEBUFFER_BLOCK_SIZE 256

#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)
inline void __getLastCudaError(const char *errorMessage, const char *file, const int line)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		FILE* log = fopen("log.txt", "a+");

		fprintf(log, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
			file, line, errorMessage, (int)err, hipGetErrorString(err));

		fclose(log);
	}
}

static float4* transformation;
static float3 camera;
static int width;
static int height;

static int* depth = NULL;
static Fragment* fragmentBuffer = NULL;

static bool backCullingEnabled = true;
static bool renderWireframe = false;

static hipArray_t framebuf_device;
static int* framebuf;
static Triangle* compactionOutput;

void Init()
{
	hipSetDevice(0);
	getLastCudaError("hipSetDevice failed");

	hipMalloc((void**)&transformation, 16 * sizeof(float));
}

RasterizerModel* CreateModel(Model* model)
{
	RasterizerModel* rasterizerModel = new RasterizerModel;

	rasterizerModel->numOfVertices = model->numOfVertices;
	rasterizerModel->numOfFaces = model->numOfFaces;

	hipMalloc((void**)&rasterizerModel->vertexBufferIn, model->numOfVertices * sizeof(VertexShaderIn));
	hipMalloc((void**)&rasterizerModel->vertexBufferOut, model->numOfVertices * sizeof(VertexShaderOut));
	hipMalloc((void**)&rasterizerModel->primitivesBuffer, model->numOfFaces * sizeof(Triangle));
	hipMalloc((void**)&rasterizerModel->indexBuffer, model->numOfFaces * 3 * sizeof(int));
	hipMalloc((void**)&compactionOutput, model->numOfFaces * sizeof(Triangle));

	for (int i = 0; i < model->numOfVertices; i++)
	{
		VertexShaderIn vertex = { model->vertices[i], model->normals[i], model->colors[i] };
		hipMemcpy(rasterizerModel->vertexBufferIn + i, &vertex, sizeof(VertexShaderIn), hipMemcpyHostToDevice);
	}

	hipMemcpy(rasterizerModel->indexBuffer, model->indices, model->numOfFaces * 3 * sizeof(int), hipMemcpyHostToDevice);

	return rasterizerModel;
}

void SetTransformation(glm::mat4x4 transf, glm::vec3 cam)
{
	hipMemcpy(transformation, &transf, sizeof(glm::mat4x4), hipMemcpyHostToDevice);

	camera = make_float3(cam.x, cam.y, cam.z);
}

void Resize(unsigned int w, unsigned int h, GLuint texture)
{
	width = w;
	height = h;

	if (depth != NULL) hipFree(depth);
	if (fragmentBuffer != NULL) hipFree(fragmentBuffer);

	hipMalloc((void**)&depth, width * height * sizeof(int));
	getLastCudaError("hipMalloc depth failed");

	hipMalloc((void**)&fragmentBuffer, width * height * sizeof(Fragment));
	getLastCudaError("hipMalloc fragmentBuffer failed");

	hipMalloc((void**)&framebuf, width * height * sizeof(int));
	getLastCudaError("hipMalloc framebuf failed");

	hipGraphicsResource* resource;

	hipGraphicsGLRegisterImage(&resource, texture, GL_TEXTURE_3D, hipGraphicsRegisterFlagsNone);
	getLastCudaError("hipGraphicsGLRegisterImage failed");

	hipGraphicsMapResources(1, &resource, 0);
	getLastCudaError("hipGraphicsMapResources failed");

	hipGraphicsSubResourceGetMappedArray(&framebuf_device, resource, 0, 0);
	getLastCudaError("hipGraphicsSubResourceGetMappedArray failed");

	hipGraphicsUnmapResources(1, &resource, 0);
	getLastCudaError("hipGraphicsUnmapResources failed");
}

void FreeRasterizer()
{
	hipFree(depth);
	hipFree(fragmentBuffer);
	hipFree(transformation);
}

void FreeModel(RasterizerModel* model)
{

}

__forceinline__ __device__ float3 transform(float4* transformation, float3 v)
{
	float w = transformation[0].w * v.x + transformation[1].w * v.y + transformation[2].w * v.z + transformation[3].w;

	return make_float3(
		(transformation[0].x * v.x + transformation[1].x * v.y + transformation[2].x * v.z + transformation[3].x) / w,
		(transformation[0].y * v.x + transformation[1].y * v.y + transformation[2].y * v.z + transformation[3].y) / w,
		(transformation[0].z * v.x + transformation[1].z * v.y + transformation[2].z * v.z + transformation[3].z) / w
	);
}

__global__ void VertexShader(const VertexShaderIn* vertexIn, VertexShaderOut* vertexOut, float4* transformation, int vertCount)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= vertCount) return;

	vertexOut[index].Pos = transform(transformation, vertexIn[index].Pos);

	vertexOut[index].Color = vertexIn[index].Color;
	vertexOut[index].ModelPos = vertexIn[index].Pos;
	vertexOut[index].Normal = vertexIn[index].Normal;
}

__global__ void Assembler(VertexShaderOut* vertexOut, Triangle* primitivesBuffer, int* indices, int facesCount, float3 camera, bool cullBackface, int width, int height)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= facesCount) return;
	Triangle* triangle = &primitivesBuffer[index];

	triangle->v1 = vertexOut[indices[index * 3]];
	triangle->v2 = vertexOut[indices[index * 3 + 1]];
	triangle->v3 = vertexOut[indices[index * 3 + 2]];

	float3 v1 = triangle->v2.ModelPos - triangle->v1.ModelPos;
	float3 v2 = triangle->v3.ModelPos - triangle->v1.ModelPos;

	triangle->Visible = !cullBackface ||
		dot(
			triangle->v1.ModelPos - camera,
			norm(cross(v1, v2))
		) > 0;
	
	triangle->minx = glm::max(glm::min(glm::min(triangle->v1.Pos.x, triangle->v2.Pos.x), triangle->v3.Pos.x), 0.f);
	triangle->miny = glm::max(glm::min(glm::min(triangle->v1.Pos.y, triangle->v2.Pos.y), triangle->v3.Pos.y), 0.f);
	triangle->maxx = glm::min(glm::max(glm::max(triangle->v1.Pos.x, triangle->v2.Pos.x), triangle->v3.Pos.x), (float)width);
	triangle->maxy = glm::min(glm::max(glm::max(triangle->v1.Pos.y, triangle->v2.Pos.y), triangle->v3.Pos.y), (float)height);

	if (triangle->minx >= triangle->maxx || triangle->miny >= triangle->maxy)
	{
		triangle->Visible = false;
	}
}

__device__ void line(float3 start, float3 end, Fragment* depthBuffer, int width, int height)
{
	float3 color = make_float3(1, 1, 1);

	int x1 = start.x;
	int y1 = start.y;

	int x2 = end.x;
	int y2 = end.y;

	int dx = abs(x2 - x1);
	int dy = abs(y2 - y1);
	int sx = (x1 < x2) ? 1 : -1;
	int sy = (y1 < y2) ? 1 : -1;
	int err = dx - dy;

	if (x1 > 0 && x1 < width && y1 > 0 && y1 < height)
	{
		depthBuffer[x1 + y1 * width].Color = color;
	}

	while (!((x1 == x2) && (y1 == y2)))
	{
		int e2 = err << 1;
		if (e2 > -dy)
		{
			err -= dy;
			x1 += sx;
		}
		if (e2 < dx)
		{
			err += dx;
			y1 += sy;
		}

		if (x1 > 0 && x1 < width && y1 > 0 && y1 < height)
		{
			depthBuffer[x1 + y1 * width].Color = color;
		}
	}
}

__global__ void RasterizeWireframe(Triangle* primitivesBuffer, int* depth, Fragment* fragmentBuffer, int width, int height, int primitivesCount)
{
	Triangle* triangle = &primitivesBuffer[blockIdx.x];
	if (!triangle->Visible) return;

	float3 start = triangle->v1.Pos;
	float3 end = triangle->v2.Pos;

	if (threadIdx.x == 1)
	{
		end = triangle->v3.Pos;
	}
	else if (threadIdx.x == 2)
	{
		start = triangle->v3.Pos;
	}

	line(start, end, fragmentBuffer, width, height);
}

__global__ void RasterizeTriangle(Triangle* primitivesBuffer, int* depth, Fragment* fragmentBuffer, int width)
{
	Triangle* triangle;

	//if (threadIdx.x == 0)
	{
		triangle = primitivesBuffer + blockIdx.x;
	}

	__shared__ float4 vals[9];
	float3 tmp = ((float3*)triangle)[threadIdx.x * 4 % 11];
	vals[threadIdx.x] = make_float4(tmp.x, tmp.y, tmp.z, 0);

	__shared__ float w[RASTERIZER_BLOCK_SIZE + 1];
	__shared__ float h[RASTERIZER_BLOCK_SIZE + 1];

	w[threadIdx.x + 1] = triangle->minx + (triangle->maxx - triangle->minx) / RASTERIZER_BLOCK_SIZE * (threadIdx.x + 1);
	h[threadIdx.y + 1] = triangle->miny + (triangle->maxy - triangle->miny) / RASTERIZER_BLOCK_SIZE * (threadIdx.y + 1);

	if (threadIdx.x == 0)
	{
		w[0] = triangle->minx;
		//vals[8] = make_float4(triangle->v3.Color.x, triangle->v3.Color.y, triangle->v3.Color.z, 0);
	}
	if (threadIdx.y == 0)
	{
		h[0] = triangle->miny;
	}

	__syncthreads();
	
	for (int y = h[threadIdx.y]; y < h[threadIdx.y + 1]; y++)
	{
		for (int x = w[threadIdx.x]; x < w[threadIdx.x + 1]; x++)
		{
			float area = 0.5f * ((vals[2].x - vals[0].x) * (vals[1].y - vals[0].y) - (vals[1].x - vals[0].x) * (vals[2].y - vals[0].y));
			
			float beta = 0.5f * ((vals[2].x - vals[0].x) * (y - vals[0].y) - (x - vals[0].x) * (vals[2].y - vals[0].y)) / area;
			float gamma = 0.5f * ((x - vals[0].x) * (vals[1].y - vals[0].y) - (vals[1].x - vals[0].x) * (y - vals[0].y)) / area;
			float alpha = 1.0f - beta - gamma;

			if (alpha >= 0.0 && alpha <= 1.0 && beta >= 0.0 && beta <= 1.0 && gamma >= 0.0 && gamma <= 1.0)
			{
				int z = (alpha * vals[0].z + beta * vals[1].z + gamma * vals[2].z) * -10000;
				int i = y * width + x;
				
				atomicMin(&depth[i], z);

				if (depth[i] == z)
				{
					Fragment* fragment = fragmentBuffer + i;

					fragment->Position.x = vals[0].x * alpha + vals[1].x * beta + vals[2].x * gamma;
					fragment->Position.y = vals[0].y * alpha + vals[1].y * beta + vals[2].y * gamma;
					fragment->Position.z = vals[0].z * alpha + vals[1].z * beta + vals[2].z * gamma;

					fragment->Normal.x = vals[3].x * alpha + vals[4].x * beta + vals[5].x * gamma;
					fragment->Normal.y = vals[3].y * alpha + vals[4].y * beta + vals[5].y * gamma;
					fragment->Normal.z = vals[3].z * alpha + vals[4].z * beta + vals[5].z * gamma;

					fragment->Color.x = vals[6].x * alpha + vals[7].x * beta + vals[8].x * gamma;
					fragment->Color.y = vals[6].y * alpha + vals[7].y * beta + vals[8].y * gamma;
					fragment->Color.z = vals[6].z * alpha + vals[7].z * beta + vals[8].z * gamma;
				}
			}
		}
	}
}

__device__ __forceinline__ int Clamp(float v)
{
	if (v < 0) v = 0;
	else if (v > 1) v = 1;
	return v * 255;
}

__global__ void CopyToFrameBuffer(Fragment* fragmentBuffer, int* backBuffer, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	//frameBuffer[i] = (Clamp(c.x) << 16) | (Clamp(c.y) << 8) | Clamp(c.z);
	int* pixel = backBuffer + y * width + x;
	Fragment* fragment = fragmentBuffer + y * width + x;

	pixel[0] = (Clamp(fragment->Color.x) << 16) | (Clamp(fragment->Color.y) << 8) | Clamp(fragment->Color.z);
	//pixel[0] = (Clamp((fragment->Normal.x + 1) / 2) << 16) | (Clamp((fragment->Normal.y + 1) / 2) << 8) | Clamp((fragment->Normal.z + 1) / 2);
}

__host__ void ClearBuffers()
{
	hipMemset(depth, 5000000, width * height * sizeof(int));
	hipMemset(fragmentBuffer, 0, width * height * sizeof(Fragment));
}

void Begin()
{
	ClearBuffers();
}

void End()
{
	CopyToFrameBuffer << <dim3((width + 16 - 1) / 16, (height + 16 - 1) / 16), dim3(16, 16) >> > (fragmentBuffer, framebuf, width, height);

	hipMemcpyToArray(framebuf_device, 0, 0, framebuf, width * height * sizeof(int), hipMemcpyDeviceToDevice);
}

void DrawModel(RasterizerModel* model)
{
	int vertexShaderGridSize = (model->numOfVertices - 1) / VERTEX_SHADER_BLOCK_SIZE + 1;
	int assemblerGridSize = (model->numOfFaces - 1) / VERTEX_SHADER_BLOCK_SIZE + 1;
	int primitiveCount = model->numOfFaces;

	VertexShader << <vertexShaderGridSize, VERTEX_SHADER_BLOCK_SIZE >> > (model->vertexBufferIn, model->vertexBufferOut, transformation, model->numOfVertices);
	Assembler << <assemblerGridSize, VERTEX_SHADER_BLOCK_SIZE >> > (model->vertexBufferOut, model->primitivesBuffer, model->indexBuffer, model->numOfFaces, camera, true, width, height);

	primitiveCount = Compact(model->numOfFaces, compactionOutput, model->primitivesBuffer);
	hipMemcpy(model->primitivesBuffer, compactionOutput, primitiveCount * sizeof(Triangle), hipMemcpyDeviceToDevice);

	RasterizeTriangle << <primitiveCount, dim3(RASTERIZER_BLOCK_SIZE, RASTERIZER_BLOCK_SIZE) >> > (model->primitivesBuffer, depth, fragmentBuffer, width);

	//RasterizeWireframe<< <numOfFaces, 3 >> > (primitivesBuffer, depth, fragmentBuffer, width, height, numOfFaces);
}


float3 operator*(float3 v, float a)
{
	return make_float3(v.x * a, v.y * a, v.z * a);
}

float3 operator/(float3 v, float a)
{
	return make_float3(v.x / a, v.y / a, v.z / a);
}

float3 operator-(float3 a, float3 b)
{
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

float3 operator+(float3 a, float3 b)
{
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

float dot(float3 a, float3 b)
{
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

float3 cross(float3 a, float3 b)
{
	return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

float length(float3 v)
{
	return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

float3 norm(float3 v)
{
	float l = length(v);

	float3 a = make_float3(v.x / l, v.y / l, v.z / l);
	return a;
}